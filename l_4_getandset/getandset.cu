
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    int gpu_count = 0;
    hipError_t error = hipGetDeviceCount(&gpu_count);

    if (error != hipSuccess || gpu_count == 0) {
        printf("You don't have any GPU!\n");
        exit(-1);
    }
    else {
        printf("The number of you GPU is %d.\n", gpu_count);
    }

    int gpu_index = 0;
    if (hipSuccess != hipSetDevice(gpu_index)) {
        printf("Fail to set GPU 0 for computing.\n");
    }
    else {
        printf("Success set GPU 0 for computing.\n");
    }
    return 0;
}