
#include <hip/hip_runtime.h>
#include <array>
#include <cstddef>
#include <iostream>

extern __shared__ float shared_array[];

__global__ void deviceToSharedMemory(float* device_array, const int device_array_size)
{
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;

    const int current_thread_id = thread_id + blockDim.x * block_id;

    if (current_thread_id < device_array_size) {
        shared_array[thread_id] = device_array[current_thread_id];
    }
    __syncthreads();

    printf("shared_memory value is %f at block idx %d\n", shared_array[thread_id], block_id);
}

int main()
{
    hipDeviceProp_t cuda_prop;
    hipGetDeviceProperties(&cuda_prop, 0);
    std::cout << "The GPU you use named: " << cuda_prop.name << std::endl;

    constexpr int array_size = 64;
    std::array<float, array_size> host_array{};
    constexpr int array_memory_size = array_size * sizeof(float);

    for (int index = 0; index < array_size; ++index) {
        host_array[index] = float(index);
    }

    float* device_array = nullptr;
    hipMalloc(&device_array, array_memory_size);
    hipMemcpy(device_array, &host_array, array_memory_size, hipMemcpyHostToDevice);

    dim3 block_dim = 32;
    dim3 grid_dim = 2;

    deviceToSharedMemory<<<grid_dim, block_dim, 32>>>(device_array, array_size);

    hipFree(device_array);
    hipDeviceReset();

    return 0;
}