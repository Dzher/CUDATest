#include <vector>
#include "../tools/deviceinit.cu"
#include "../tools/errorcheck.cu"

void memcpyWithCheck()
{
    getAndSetGpu0();

    // set host float memory
    std::vector<float> float_mem(4, 0.0);

    // set device memory
    float* device_float_mem;
    hipError_t error = cudaErrorCheck(hipMalloc(&device_float_mem, 4), __FILE__, __LINE__);
    hipMemset(device_float_mem, 0, 4);

    // error hipMemcpyKind value here
    cudaErrorCheck(hipMemcpy(device_float_mem, &float_mem, 4, hipMemcpyDeviceToHost), __FILE__, __LINE__);

    cudaErrorCheck(hipFree(device_float_mem), __FILE__, __LINE__);

    cudaErrorCheck(hipDeviceReset(), __FILE__, __LINE__);

    return;
}

int main()
{
    memcpyWithCheck();
    return 0;
}