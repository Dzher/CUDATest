
#ifndef __CUDA_ERROR_CHECK__FUNC__
#define __CUDA_ERROR_CHECK__FUNC__

#include <stdio.h>

hipError_t cudaErrorCheck(hipError_t error_code, const char* file_name, int error_line)
{
    if (error_code != hipSuccess) {
        printf("CUDA Error: \ncode=%d, name=%s, description=%s\nfile=%s, line%d\n", error_code,
               hipGetErrorName(error_code), hipGetErrorString(error_code), file_name, error_line);
        return error_code;
    }
    return error_code;
}

#endif  //__CUDA_ERROR_CHECK__FUNC__
