
#include <hip/hip_runtime.h>
#include <stdio.h>

void setGPU()
{
    int gpu_count = 0;
    hipError_t error = hipGetDeviceCount(&gpu_count);

    if (error != hipSuccess || gpu_count == 0) {
        printf("You don't have any GPU!\n");
        exit(-1);
    }
    else {
        printf("The number of you GPU is %d.\n", gpu_count);
    }

    int gpu_index = 0;
    if (hipSuccess != hipSetDevice(gpu_index)) {
        printf("Fail to set GPU 0 for computing.\n");
    }
    else {
        printf("Success set GPU 0 for computing.\n");
    }
}

__device__ float add(const float x, const float y)
{
    return x + y;
}

__global__ void addFromGpu(float* mem_a, float* mem_b, float* result, const int adder_size)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < adder_size) {
        result[id] = add(mem_a[id], mem_b[id]);
    }
}

void randomInitialFloatData(float* mem, int count)
{
    for (int i = 0; i < count; ++i) {
        mem[i] = (float)(rand() / 10000.f);
    }
}

void memHostDeviceCpy()
{
    int mem_size = 512;
    size_t byte_size = mem_size * sizeof(float);

    float* host_float_a = new float[byte_size];
    float* host_float_b = new float[byte_size];
    float* host_float_result = new float[byte_size];

    if (host_float_a && host_float_b && host_float_result) {
        memset(host_float_a, 0, byte_size);
        memset(host_float_b, 0, byte_size);
        memset(host_float_result, 0, byte_size);
    }
    else {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }

    float* device_float_a;
    float* device_float_b;
    float* device_float_result;

    if (hipSuccess == hipMalloc(&device_float_a, byte_size) &&
        hipSuccess == hipMalloc(&device_float_b, byte_size) &&
        hipSuccess == hipMalloc(&device_float_result, byte_size)) {
        hipMemset(device_float_a, 0, byte_size);
        hipMemset(device_float_b, 0, byte_size);
        hipMemset(device_float_result, 0, byte_size);
    }
    else {
        printf("Fail to allocate memory!\n");
        free(host_float_a);
        free(host_float_b);
        free(host_float_result);
        exit(-1);
    }

    randomInitialFloatData(host_float_a, mem_size);
    randomInitialFloatData(host_float_b, mem_size);

    hipMemcpy(device_float_a, host_float_a, byte_size, hipMemcpyHostToDevice);
    hipMemcpy(device_float_b, host_float_b, byte_size, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid((mem_size + block.x - 1) / 32);

    addFromGpu<<<grid, block>>>(device_float_a, device_float_b, device_float_result, mem_size);
    hipDeviceSynchronize();

    hipMemcpy(host_float_result, device_float_result, byte_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        printf("id=%d\tmatrix_a:%.4f\tmatrix_b:%.4f\tresult=%.4f\n", i + 1, host_float_a[i], host_float_b[i],
               host_float_result[i]);
    }

    free(host_float_a);
    free(host_float_b);
    free(host_float_result);
    hipFree(device_float_a);
    hipFree(device_float_b);
    hipFree(device_float_result);

    hipDeviceReset();
    return;
}

int main()
{
    setGPU();
    memHostDeviceCpy();

    return 0;
}