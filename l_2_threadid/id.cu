
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printThreadId()
{
    const auto block_id = blockIdx.x;
    const auto thread_id = threadIdx.x;
    const int id = threadIdx.x + blockIdx.x * blockDim.x;

    printf("Hello from block %d and thread %d, global id %d\n", block_id, thread_id, id);
}

int main()
{
    printThreadId<<<2, 4>>>();
    hipDeviceSynchronize();

    return 0;
}