
#include <hip/hip_runtime.h>
#include <array>
#include <iostream>

__device__ int device_var = 1234;
__device__ int device_array[2];

__global__ void devicePrint()
{
    device_array[0] += device_var;
    device_array[1] += device_var;
    printf("device var = %d, device array[0] = %d, device array[1] = %d\n", device_var, device_array[0],
           device_array[1]);
}

int main()
{
    hipDeviceProp_t cuda_prop;
    hipGetDeviceProperties(&cuda_prop, 0);
    std::cout << "The GPU name is " << cuda_prop.name << std::endl;

    std::array<int, 2> host_array{0, 0};
    hipMemcpyToSymbol(HIP_SYMBOL(device_array), &host_array, sizeof(int) * host_array.size());

    dim3 cuda_grid{1};
    dim3 cuda_block{1};

    devicePrint<<<cuda_grid, cuda_block>>>();
    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&host_array, HIP_SYMBOL(device_array), sizeof(int) * host_array.size());

    std::cout << "host array[0] = " << host_array[0] << ","
              << " host array[1] = " << host_array[1] << std::endl;

    hipDeviceReset();
    return 0;
}