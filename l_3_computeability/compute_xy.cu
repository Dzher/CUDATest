
#include <hip/hip_runtime.h>
#include <stdio.h>

void printfGPUComputeXY()
{
    int gpu_count = 0;
    hipGetDeviceCount(&gpu_count);
    if (gpu_count > 0) {
        hipDeviceProp_t cuda_device_prop;
        for (int gpu_index = 0; gpu_index < gpu_count; ++gpu_index) {
            hipGetDeviceProperties(&cuda_device_prop, gpu_index);
            printf("Your GPU Compute Ability is: %d.%d\n", cuda_device_prop.major, cuda_device_prop.minor);
        }
    }
}

int main()
{
    printfGPUComputeXY();
}