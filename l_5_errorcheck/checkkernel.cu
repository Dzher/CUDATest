#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../tools/errorcheck.cu"

__global__ void testFromGpu()
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    printf("Test from the %d thread\n", id);
}

int main()
{
    dim3 block(1025);
    dim3 grid(1);

    testFromGpu<<<grid, block>>>();
    cudaErrorCheck(hipGetLastError(), __FILE__, __LINE__);
    cudaErrorCheck(hipDeviceSynchronize(), __FILE__, __LINE__);

    hipDeviceSynchronize();
}