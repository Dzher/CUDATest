#ifndef __CUDA__DEVICE_SET_AND_GET__
#define __CUDA__DEVICE_SET_AND_GET__

#include "stdio.h"

void getAndSetGpu0()
{
    int gpu_count;
    if (hipSuccess != hipGetDeviceCount(&gpu_count)) {
        printf("Error: Cannot get your device count!\n");
        return;
    }
    else {
        printf("Get GPU Count Success with number:%d\n", gpu_count);
    }

    if (hipSuccess != hipSetDevice(0)) {
        printf("Error: Cannot initial your device!\n");
        return;
    }
    else {
        printf("Initial GPU 0 Success!\n");
    }
}

#endif  //__CUDA__DEVICE_SET_AND_GET__