
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGpu()
{
    printf("Hello world from the GPU\n");
}

int main()
{
    helloFromGpu<<<4, 4>>>();
    hipDeviceSynchronize();

    return 0;
}